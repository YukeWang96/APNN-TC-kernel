#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------
// File: alexnet.cu
// VGG-16 BNN inference source file for ImageNet. 
// ---------------------------------------------------------------------------
// See our arXiv paper for detail: https://arxiv.org/abs/2006.16578
// Ang Li, Scientist, Pacific Northwest National Laboratory(PNNL), U.S.
// Homepage: http://www.angliphd.com
// GitHub repo: http://www.github.com/pnnl/TCBNN
// PNNL-IPID: 31925-E, ECCN: EAR99, IR: PNNL-SA-152850
// BSD Lincese.
// Richland, 99352, WA, USA. June-30-2020.
// ---------------------------------------------------------------------------

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <iostream>
#include <string>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <fstream>
#include <vector>
#include "utility.h"
#include "param.h"
#include "kernel.cuh"
#include "data.h"

#define NUM_PROF 100

#define w1_a2
// #define w2_a2
// #define w1_a4

#ifdef w1_a2
const unsigned w_bit = 1;
const unsigned act_bit = 2;
#include "51_APCONV_w1a2_t64.cu"           // 13.978
// #include "52_APCONV_w1a2_t128.cu"             // 12.321

#include "31_APMM_w1a2_t128.cu" 
#endif

#ifdef w2_a2
const unsigned w_bit = 2;
const unsigned act_bit = 2;
#include "48_APCONV_w2a2_t64.cu"             // 
// #include "49_APCONV_w2a2_t128.cu"         //
// #include "50_APCONV_w2a2_t128.cu"        //

#include "13_APMM_w2a2_t64.cu"  //0.013 ms
// #include "29_APMM_w2a2_t128.cu"    //0.018 ms
// #include "30_APMM_w2a2_t128.cu" //0.008 ms
#endif

#ifdef w1_a4
const unsigned w_bit = 1;
const unsigned act_bit = 4;
#include "53_APCONV_w1a4_t64.cu"            // 
// #include "54_APCONV_w1a4_t128.cu"        // 

#include "33_APMM_w1a4_t64.cu"      
// #include "32_APMM_w1a4_t128.cu"           
#endif

#ifdef w1_a8
const unsigned w_bit = 1;
const unsigned act_bit = 8;
// #include "55_APCONV_w1a8_t64.cu"        //  ms
// #include "56_APCONV_w1a8_t128.cu"        // ms
#endif


using namespace cooperative_groups;
using namespace std;

__global__ void vggnet128(
        InConv128LayerParam* bconv1, 
        Conv128LayerParam* bconv2, 
        Conv128LayerParam* bconv3,
        Conv128LayerParam* bconv4, 
        Conv128LayerParam* bconv5, 
        Conv128LayerParam* bconv6,
        Conv128LayerParam* bconv7, 
        Conv128LayerParam* bconv8,
        Conv128LayerParam* bconv9, 
        Conv128LayerParam* bconv10, 
        Conv128LayerParam* bconv11,
        Conv128LayerParam* bconv12,
        Conv128LayerParam* bconv13,
        Fc128LayerParam* bfc1, 
        Fc128LayerParam* bfc2, 
        Out128LayerParam* bout)
{
    grid_group grid = this_grid();
    
    //========= Conv1 ============
    InConv128Layer(bconv1);
    grid.sync();
    //========= Conv2 ============
    Conv128Layer_new(bconv2);
    grid.sync();
    //========= Conv3 ============
    Conv128Layer_new(bconv3);
    grid.sync();
    //========= Conv4 ============
    Conv128Layer_new(bconv4);
    grid.sync();
    //========= Conv5 ============
    Conv128Layer_new(bconv5);
    grid.sync();
    //========= Conv6 ============
    Conv128Layer_new(bconv6);
    grid.sync();
    //========= Conv7 ============
    Conv128Layer_new(bconv7);
    grid.sync();
    //========= Conv8 ============
    Conv128Layer_new(bconv8);
    grid.sync();
    //========= Conv9 ============
    Conv128Layer_new(bconv9);
    grid.sync();
    //========= Conv10 ============
    Conv128Layer_new(bconv10);
    grid.sync();
    //========= Conv11 ============
    Conv128Layer_new(bconv11);
    grid.sync();
    //========= Conv12 ============
    Conv128Layer_new(bconv12);
    grid.sync();
    //========= Conv13 ============
    Conv128Layer_new(bconv13);
    grid.sync();
    //========= Fc1 ============
    Fc128Layer_new(bfc1);
    grid.sync();
    //========= Fc2 ============
    Fc128Layer_new(bfc2);
    grid.sync();
    ////========== Output ===========
    Out128Layer_new(bout);
}
  
     
int main()
{
    int dev = 0;
    hipSetDevice(dev);

    const unsigned batch = 128;
    const unsigned output_size = 1000;
    const unsigned image_height = 224;
    const unsigned image_width = 224;
    const unsigned image_channel = 3;
    const unsigned filter_height = 3;
    const unsigned filter_width = 3;
    const unsigned n_hidden = 4096;

    //=============== Get Input and Label =================
    float* images = (float*)malloc(batch*image_height*image_width*image_channel*sizeof(float));
    unsigned* image_labels = (unsigned*)malloc(batch*sizeof(unsigned));
//     read_ImageNet_normalized("./imagenet_files.txt", images, image_labels, batch);
    
    //================ Get Weight =================
    FILE* config_file = fopen("./vgg_imagenet.csv","r");
    //================ Set Network =================
    //Bconv1 Layer
    InConv128LayerParam* bconv1 = new InConv128LayerParam("Conv1", image_height, image_width, 
            filter_height, filter_width, 3, 64, batch, act_bit, w_bit); 
    InConv128LayerParam* bconv1_gpu = bconv1->initialize(images, config_file);
    //Bconv2 Layer
    Conv128LayerParam* bconv2 = new Conv128LayerParam("Conv2", bconv1->output_height, 
            bconv1->output_width, filter_height, filter_width, 64, 64, batch, act_bit, w_bit, 1, 1,
            true, 2, 2, false);

    Conv128LayerParam* bconv2_gpu = bconv2->initialize(config_file, bconv1->get_output_gpu());
    //Bconv3 Layer
    Conv128LayerParam* bconv3 = new Conv128LayerParam("Conv3", bconv2->output_height, 
            bconv2->output_width, filter_height, filter_width, 64, 128, batch, act_bit, w_bit);
    Conv128LayerParam* bconv3_gpu = bconv3->initialize(config_file, bconv2->get_output_gpu());
    //Bconv4 Layer
    Conv128LayerParam* bconv4 = new Conv128LayerParam("Conv4", bconv3->output_height, 
            bconv3->output_width, filter_height, filter_width, 128, 128, batch, act_bit, w_bit, 1, 1,
            true, 2, 2, false);
    Conv128LayerParam* bconv4_gpu = bconv4->initialize(config_file, bconv3->get_output_gpu());
    //Bconv5 Layer
    Conv128LayerParam* bconv5 = new Conv128LayerParam("Conv5", bconv4->output_height, 
            bconv4->output_width, filter_height, filter_width, 128, 256, batch, act_bit, w_bit);
    Conv128LayerParam* bconv5_gpu = bconv5->initialize(config_file, bconv4->get_output_gpu());
    //Bconv6 Layer
    Conv128LayerParam* bconv6 = new Conv128LayerParam("Conv6", bconv5->output_height, 
            bconv5->output_width, filter_height, filter_width, 256, 256, batch, act_bit, w_bit);
    Conv128LayerParam* bconv6_gpu = bconv6->initialize(config_file, bconv5->get_output_gpu());
    //Bconv7 Layer
    Conv128LayerParam* bconv7 = new Conv128LayerParam("Conv7", bconv6->output_height, 
            bconv6->output_width, filter_height, filter_width, 256, 256, batch, act_bit, w_bit, 1, 1,
            true, 2, 2, false);
    Conv128LayerParam* bconv7_gpu = bconv7->initialize(config_file, bconv6->get_output_gpu());
    //Bconv8 Layer
    Conv128LayerParam* bconv8 = new Conv128LayerParam("Conv8", bconv7->output_height, 
            bconv7->output_width, filter_height, filter_width, 256, 512, batch, act_bit, w_bit);
    Conv128LayerParam* bconv8_gpu = bconv8->initialize(config_file, bconv7->get_output_gpu());
    //Bconv9 Layer
    Conv128LayerParam* bconv9 = new Conv128LayerParam("Conv9", bconv8->output_height, 
            bconv8->output_width, filter_height, filter_width, 512, 512, batch, act_bit, w_bit);
    Conv128LayerParam* bconv9_gpu = bconv9->initialize(config_file, bconv8->get_output_gpu());
    //Bconv10 Layer
    Conv128LayerParam* bconv10 = new Conv128LayerParam("Conv10", bconv9->output_height, 
            bconv9->output_width, filter_height, filter_width, 512, 512, batch, act_bit, w_bit, 1, 1,
            true, 2, 2, false);
    Conv128LayerParam* bconv10_gpu = bconv10->initialize(config_file, bconv9->get_output_gpu());
    //Bconv11 Layer
    Conv128LayerParam* bconv11 = new Conv128LayerParam("Conv11", bconv10->output_height, 
            bconv10->output_width, filter_height, filter_width, 512, 512, batch, act_bit, w_bit);
    Conv128LayerParam* bconv11_gpu = bconv11->initialize(config_file, bconv10->get_output_gpu());
    //Bconv12 Layer
    Conv128LayerParam* bconv12 = new Conv128LayerParam("Conv12", bconv11->output_height, 
            bconv11->output_width, filter_height, filter_width, 512, 512, batch, act_bit, w_bit);
    Conv128LayerParam* bconv12_gpu = bconv12->initialize(config_file, bconv11->get_output_gpu());
    //Bconv13 Layer
    Conv128LayerParam* bconv13 = new Conv128LayerParam("Conv13", bconv12->output_height, 
            bconv12->output_width, filter_height, filter_width, 512, 512, batch, act_bit, w_bit, 1, 1,
            true, 2, 2, true);
    Conv128LayerParam* bconv13_gpu = bconv13->initialize(config_file, bconv12->get_output_gpu());
    //Fc1 Layer
    Fc128LayerParam* bfc1 = new Fc128LayerParam("Fc1", batch, (bconv13->output_height)
            *(bconv13->output_width)*512, n_hidden, act_bit, w_bit); 
    Fc128LayerParam* bfc1_gpu = bfc1->initialize(config_file, bconv13->get_output_gpu());
    //Fc2 Layer
    Fc128LayerParam* bfc2 = new Fc128LayerParam("Fc2", batch, n_hidden, n_hidden, act_bit, w_bit); 
    Fc128LayerParam* bfc2_gpu = bfc2->initialize(config_file, bfc1->get_output_gpu());
    //Out Layer
    Out128LayerParam* bout = new Out128LayerParam("Fout", batch, n_hidden, output_size, act_bit, w_bit);
    Out128LayerParam* bout_gpu = bout->initialize(config_file, bfc2->get_output_gpu());  

    //================ Setup Kernel =================
    int numThreads = 128;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    int numBlocksPerSm;
    int shared_memory = 512*sizeof(int)*32;

    hipFuncSetAttribute(reinterpret_cast<const void*>(vggnet128), hipFuncAttributeMaxDynamicSharedMemorySize,shared_memory);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, vggnet128, numThreads, shared_memory);

    void* args[] = {&bconv1_gpu, &bconv2_gpu, &bconv3_gpu, &bconv4_gpu, &bconv5_gpu, &bconv6_gpu,
        &bconv7_gpu, &bconv8_gpu, &bconv9_gpu, &bconv10_gpu, &bconv11_gpu, &bconv12_gpu, &bconv13_gpu,
        &bfc1_gpu, &bfc2_gpu, &bout_gpu};

    START_TIMER;
    for (int i=0; i<NUM_PROF; i++)
    hipLaunchCooperativeKernel((void*)vggnet128, numBlocksPerSm*deviceProp.multiProcessorCount, 
            numThreads, args, shared_memory);
     CUDA_CHECK_KERNEL();

    STOP_TIMER;
    printf("Time: %.3f (ms)\n", milliseconds/NUM_PROF);

//     float* output = bout->download_output();
    //validate_prediction(output, image_labels, output_size, batch);

    /*
    float* out = bconv2->download_full_output();
    for (int i=0; i<512; i++)
    //for (int i=4096; i<4096+512; i++)
    {
        printf("%.f ", out[i]);
        if ((i+1)%32==0) printf("\n");
    }
    printf("\n===%f===\n", bout->bn_scale[0]);
    */

    delete bconv1;
    delete bconv2;
    delete bconv3;
    delete bconv4;
    delete bconv5;
    delete bconv6;
    delete bconv7;
    delete bconv8;
    delete bconv9;
    delete bconv10;
    delete bconv11;
    delete bconv12;
    delete bconv13;
    delete bfc1;
    delete bfc2;
    delete bout;

    return 0;

}















