#include "hip/hip_runtime.h"
/*
  1-bit BMMA code.
  Runs at 500TOPS for matrix size of 4096x4096x8192.
  Borrows largely from CUDA-SDK.

  By Boyuan
*/
#ifndef NEW_KERNEL_CONV
#define NEW_KERNEL_CONV

#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "param.h"

#define CHUNK_K 4
#define SKEW 0 
#define WARPS_PER_BLOCK 8
#define WARP_SIZE 32
#define THREADS_PER_BLOCK WARP_SIZE*WARPS_PER_BLOCK
#define CHUNK_LINE_BYTES CHUNK_K*sizeof(int4)
#define WARP_COPY_BYTES WARP_SIZE*sizeof(int4)
#define CHUNK_COPY_LINES_PER_WARP WARP_COPY_BYTES / CHUNK_LINE_BYTES
#define CHUNK_COPY_LINE_LANES WARP_SIZE / CHUNK_COPY_LINES_PER_WARP
#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4
#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2
#define BLOCK_ROW_TILES WARP_ROW_TILES*BLOCK_ROW_WARPS
#define BLOCK_COL_TILES WARP_COL_TILES*BLOCK_COL_WARPS
#define M 8
#define N 8
#define K 128



using namespace nvcuda;
using namespace nvcuda::wmma::experimental;

// Assume that Kernel size is 3x3.
// Assume CIN is 128.
__inline__  __device__
void Conv128Layer_new(Conv128LayerParam* p) {

  const int4 *W = (int4*) (p->filter_gpu);
  const int4 *X = (int4*) (p->input_gpu);
  int *Output = (int*) p->output_gpu;
  const int Height = p->input_height;
  const int Width =  p->input_width;
  const int CIN = p->input_channels;
  const int COUT =  p->output_channels;

  // GEMM Configuration
  int X_bit_offset = Height * Width * CIN/128;

  extern __shared__ int4 shmem[][CHUNK_K+SKEW]; // TODO: Padding opportunity may exist here.

  // Warp and lane identification.
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;

  for (unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
    const unsigned int block_i = (block_pos/(COUT/64)) / (Width/8) * 4;
    const unsigned int block_j = (block_pos/(COUT/64)) % (Width/8) * 8;
    const unsigned int block_z = block_pos % (COUT/64) * 64;

    if (block_i >= Height) {
      break;
    }

    int image_starting_idx = block_i * Width * CIN/128 + block_j * CIN/128;

    wmma::fragment<wmma::accumulator, 8, 8, 128, int> c[WARP_COL_TILES]
                                                     [WARP_ROW_TILES];

    for(int i=0; i < WARP_COL_TILES; i++)
      for(int j = 0; j < WARP_ROW_TILES; j++)
        wmma::fill_fragment(c[i][j], 0);
    
    if (threadIdx.x < 120) {
      int threadPart = threadIdx.x/60;
      int threadOffset = threadIdx.x%60;
      int GL_idx = threadPart * X_bit_offset + (threadOffset/10)*Width + threadOffset%10 + image_starting_idx;
      *(&shmem[128][0]+threadIdx.x) = X[GL_idx];
    }
    __syncthreads();

    // Go through the global K dimension by a fixed step at a time.
#pragma unroll
    for (int tile_k = 0; tile_k < int(9*CIN/128/4); tile_k += CHUNK_K) {

      int SHMEM_i = threadIdx.x/4;
      int SHMEM_part = SHMEM_i / 32;
      int SHMEM_offset = SHMEM_i % 32;
      int feature_expand_idx = SHMEM_part * 15 * CIN/2 + (SHMEM_offset/8)*10*CIN/128 + (SHMEM_offset%8)*CIN/128;

      int t = threadIdx.x % 4;
      int thread_expand_idx = feature_expand_idx + (tile_k*4+t)/(3*CIN/128)*10*(CIN/128) + (tile_k*4+t)%(3*CIN/128);
      shmem[SHMEM_i][t] = *(&shmem[128][0]+thread_expand_idx);

      SHMEM_i += 64;
      int weight_load_idx = SHMEM_part * 9 * CIN * COUT / 128 + (block_z + SHMEM_offset) * 9 * CIN/128;
      int thread_load_idx = weight_load_idx + (tile_k*4 + t) * CIN/128;
      shmem[SHMEM_i][t] = W[thread_load_idx];

      __syncthreads();

      // Compute a grid of C matrix tiles in each warp.
#pragma unroll
      for (int k_step = 0; k_step < CHUNK_K; k_step++) {
        wmma::fragment<wmma::matrix_a, M, N, K, precision::b1, wmma::row_major> a[WARP_COL_TILES];
        wmma::fragment<wmma::matrix_b, M, N, K, precision::b1, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
          size_t shmem_idx_a = (warpId / 2) * M * 4 + (i * M);
          const int4 *tile_ptr = &shmem[shmem_idx_a][k_step];

          wmma::load_matrix_sync(a[i], tile_ptr, (CHUNK_K + SKEW)*128);

#pragma unroll
          for (int j = 0; j < WARP_ROW_TILES; j++) {
            if (i == 0) {
              // Load the B matrix fragment once, because it is going to be
              // reused against the other A matrix fragments.
              size_t shmem_idx_b = 64 +
                                   (WARP_ROW_TILES * N) * (warpId % 2) +
                                   (j * N);
              const int4 *tile_ptr = &shmem[shmem_idx_b][k_step * (K/128)];

              wmma::load_matrix_sync(b[j], tile_ptr, (CHUNK_K + SKEW)*128);
            }
            // printf("ckpt4\n");

            wmma::bmma_sync(c[i][j], a[i], b[j], c[i][j]);
          }
        }
      }
      __syncthreads();
    }

    // Needs special handle for the remaining K.

    // Store the D fragments to shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
        int *tile_ptr = (int*)&shmem[0][0] + warpId*8*64 + (i*4+j) * 64;
        wmma::store_matrix_sync(tile_ptr, c[i][j], 8,  wmma::mem_row_major);
      }
    }

    __syncthreads();

    if (threadIdx.x < 32) {
      int num1 = 0;
      int num2 = 0;
      for (int j = 0; j < 32; j++) {
        int tile_i = threadIdx.x%16/8;
        int element_i = (threadIdx.x%16)%8;  
        int tile_j = j%32/8;
        int element_j = (j%32)%8;
        int final_i = warpId * 8 + tile_i*4+tile_j;
        int final_j = element_i *8 + element_j;
        int v0 = *((int*)&shmem[0][0]+final_i*64+final_j);
        int v1 = *((int*)&shmem[0][0]+final_i*64+final_j+32);
        int v2 = *((int*)&shmem[0][0]+(final_i+32)*64+final_j);
        int v3 = *((int*)&shmem[0][0]+(final_i+32)*64+final_j+32);
        int tmp = v0 + 2*v1 + 2*v2 + 4*v3;
        int tmp1 = tmp&1;
        int tmp2 = tmp&2;
        num1 = (num1 << 1) | tmp1;
        num2 = (num2 << 1) | tmp2;
      }
      *(Output+(threadIdx.x/8)*Width + threadIdx.x%8) = num1;
      *(Output+(threadIdx.x/8)*Width + threadIdx.x%8+ Height*Width*COUT/32) = num2;
    }

    __syncthreads();
  }
}
#endif