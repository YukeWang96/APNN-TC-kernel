/***************************************************************************************************
 * Copyright (c) 2017-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
Please check example 07 and 08 for the basics of tensor op gemm kernels.  On NVIDIA Ampere
architecture, most concept still holds.  The two main differences are

1. NVIDIA Ampere architecture introduces a new series of tensor core instructions (see 
   include/cutlass/arch/mma_sm80.h) which are more efficient on Ampere.

2. NVIDIA Ampere architecture uses cp_async() to build multistage software pipeline to better hide
   latency (see include/cutlass/gemm/threadblock/mma_multistage.h)

Moreover, NVIDIA Ampere architecture starts supporting tfloat32 (see include/cutlass/tfloat32.h)
data types in tensor cores.  One big advantage is that we can load in fp32 data and convert them
implicitly to tf32 inside the GEMM kernel which means no change is needed to accelerate traditional
fp32 data by using NVIDIA Ampere architecture.
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"
#include "config.h"

int run(int M, int N, int K) {

  int length_m = M;
  int length_n = N;
  int length_k = K;

  // Create a tuple of problem size for matrix multiplication
  cutlass::gemm::GemmCoord problem_size(length_m, length_n, length_k);

  // Initialize tensors using CUTLASS helper functions
  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(problem_size.mk());  // <- Create matrix A with dimensions M x K
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(problem_size.kn());  // <- Create matrix B with dimensions K x N
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(problem_size.mn());  // <- Create matrix C with dimensions M x N
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
  
  // CUTLASS kernel Reference
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // reference kernel

  // Copy data from host to GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_c.sync_device();
  tensor_d.sync_device();
  tensor_ref_d.sync_device();

  // Initialize alpha and beta for dot product computation
  ElementComputeEpilogue alpha = ElementComputeEpilogue(1);
  ElementComputeEpilogue beta = ElementComputeEpilogue(0);

  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Create a tuple of gemm kernel arguments. This is later passed as arguments to launch
  // instantiated CUTLASS kernel
  typename Gemm::Arguments arguments{problem_size,  // <- problem size of matrix multiplication
                                     tensor_a.device_ref(),  // <- reference to matrix A on device
                                     tensor_b.device_ref(),  // <- reference to matrix B on device
                                     tensor_c.device_ref(),  // <- reference to matrix C on device
                                     tensor_d.device_ref(),  // <- reference to matrix D on device
                                     {alpha, beta},          // <- tuple of alpha and beta
                                     split_k_slices};        // <- k-dimension split factor

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm_op;

  // Initialize CUTLASS kernel with arguments and workspace pointer
  cutlass::Status status = gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(status);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  // Launch initialized CUTLASS kernel
  for(int trial = 0; trial < NUM_PROFILE; trial++) {
    // printf("[%d]\n", trial);
    status = gemm_op();
    CUTLASS_CHECK(status);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("CUTLASS-GEMM (%d-bit). M: %6d, N: %6d, K: %6d,\tTOPS: %4.2f\tTime (ms): %.3f\n", BIT_WIDTH, M, N, K, 
                                                static_cast<double>(NUM_PROFILE*(static_cast<double>(M) * N * K * 2) /
                                               (milliseconds / 1000.)) / 1e12, milliseconds/NUM_PROFILE);
    /*
  // Create instantiation for device reference gemm kernel
  cutlass::reference::device::Gemm<ElementInputA,
                                   LayoutInputA,
                                   ElementInputB,
                                   LayoutInputB,
                                   ElementOutput,
                                   LayoutOutput,
                                   ElementComputeEpilogue,
                                   ElementComputeEpilogue> gemm_device;

  // Launch device reference gemm kernel
  gemm_device(problem_size,
              alpha,
              tensor_a.device_ref(),
              tensor_b.device_ref(),
              beta,
              tensor_c.device_ref(),
              tensor_ref_d.device_ref());

  // Wait for kernels to finish
  hipDeviceSynchronize();

  // Copy output data from CUTLASS and reference kernel to host for comparison
  tensor_d.sync_host();
  tensor_ref_d.sync_host();

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  bool passed = cutlass::reference::host::TensorEquals(
    tensor_d.host_view(),
    tensor_ref_d.host_view());

  std::cout << (passed ? "Passed" : "Failed") << std::endl;

  // return (passed ? 0  : -1);*/
  return 0;
}

int main(int argc, char* argv[]) {

  if (argc < 2){
    printf("Usage: ./prog M N K\n");
    return -1;
  }

  int M = atoi(argv[1]);
  int N = atoi(argv[2]);
  int K = atoi(argv[3]);


  bool notSupported = false;

  // Ampere Tensor Core operations exposed with mma.sync and ldmatrix are first available
  // in CUDA 11.0. 
  //
  // CUTLASS must be compiled with CUDA 11.0 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ >= 11)) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (!((props.major * 10 + props.minor) >= 80)) {
    std::cerr << "Turing Tensor Core operations must be run on a machine with compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  return run(M, N, K);
}