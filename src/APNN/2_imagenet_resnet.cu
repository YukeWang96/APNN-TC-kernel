#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------
// File: imagenet_resnet.cu
// ResNet-18 BNN inference source file for ImageNet. 
// ---------------------------------------------------------------------------
// See our arXiv paper for detail: https://arxiv.org/abs/2006.16578
// Ang Li, Scientist, Pacific Northwest National Laboratory(PNNL), U.S.
// Homepage: http://www.angliphd.com
// GitHub repo: http://www.github.com/pnnl/TCBNN
// PNNL-IPID: 31925-E, ECCN: EAR99, IR: PNNL-SA-152850
// BSD Lincese.
// Richland, 99352, WA, USA. June-30-2020.
// ---------------------------------------------------------------------------

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <iostream>
#include <string>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <fstream>
#include <vector>
#include "utility.h"
#include "param.h"
#include "kernel.cuh"
#include "data.h"

#define NUM_PROF 100

// #define w1_a2
#define w2_a2
// #define w1_a4

#ifdef w1_a2
const unsigned w_bit = 1;
const unsigned act_bit = 2;
#include "51_APCONV_w1a2_t64.cu"           // 0.152 ms
// #include "52_APCONV_w1a2_t128.cu"       // 

#include "31_APMM_w1a2_t128.cu" 
#endif

#ifdef w2_a2
const unsigned w_bit = 2;
const unsigned act_bit = 2;
#include "48_APCONV_w2a2_t64.cu"             // 0.209 ms
// #include "49_APCONV_w2a2_t128.cu"         //
// #include "50_APCONV_w2a2_t128.cu"        //

#include "13_APMM_w2a2_t64.cu"  //0.013 ms
// #include "29_APMM_w2a2_t128.cu"    //0.018 ms
// #include "30_APMM_w2a2_t128.cu" //0.008 ms
#endif

#ifdef w1_a4
const unsigned w_bit = 1;
const unsigned act_bit = 4;
#include "53_APCONV_w1a4_t64.cu"            // 
// #include "54_APCONV_w1a4_t128.cu"        // 

#include "33_APMM_w1a4_t64.cu"      
// #include "32_APMM_w1a4_t128.cu"           
#endif

#ifdef w1_a8
const unsigned w_bit = 1;
const unsigned act_bit = 8;
// #include "55_APCONV_w1a8_t64.cu"        //  ms
// #include "56_APCONV_w1a8_t128.cu"        // ms
#endif


using namespace cooperative_groups;
using namespace std;

__global__ void resnet128(
        InConv128LayerParam* bconv1, 
        Conv128LayerParam* l1b1c1, 
        Conv128LayerParam* l1b1c2,
        Conv128LayerParam* l1b2c1, 
        Conv128LayerParam* l1b2c2,
        Conv128LayerParam* l2b1c1, 
        Conv128LayerParam* l2b1c2,
        Conv128LayerParam* l2b2c1, 
        Conv128LayerParam* l2b2c2,
        Conv128LayerParam* l3b1c1, 
        Conv128LayerParam* l3b1c2,
        Conv128LayerParam* l3b2c1, 
        Conv128LayerParam* l3b2c2,
        Conv128LayerParam* l4b1c1, 
        Conv128LayerParam* l4b1c2,
        Conv128LayerParam* l4b2c1, 
        Conv128LayerParam* l4b2c2,
        Fc128LayerParam* bfc1, 
        Out128LayerParam* bout)
{
    grid_group grid = this_grid();
    //========= Conv1 ============
    InConv128Layer(bconv1);
    grid.sync();
    //========= L1B1 ============
    Conv128Layer_new(l1b1c1);
    grid.sync();
    Conv128Layer_new(l1b1c2);
    grid.sync();
    //========= L1B2 ============
    Conv128Layer_new(l1b2c1);
    grid.sync();
    Conv128Layer_new(l1b2c2);
    grid.sync();
    //========= L2B1 ============
    Conv128Layer_new(l2b1c1);
    grid.sync();
    Conv128Layer_new(l2b1c2);
    grid.sync();
    //========= L2B2 ============
    Conv128Layer_new(l2b2c1);
    grid.sync();
    Conv128Layer_new(l2b2c2);
    grid.sync();
    //========= L3B1 ============
    Conv128Layer_new(l3b1c1);
    grid.sync();
    Conv128Layer_new(l3b1c2);
    grid.sync();
    //========= L3B2 ============
    Conv128Layer_new(l3b2c1);
    grid.sync();
    Conv128Layer_new(l3b2c2);
    grid.sync();
    //========= L4B1 ============
    Conv128Layer_new(l4b1c1);
    grid.sync();
    Conv128Layer_new(l4b1c2);
    grid.sync();
    //========= L4B2 ============
    Conv128Layer_new(l4b2c1);
    grid.sync();
    Conv128Layer_new(l4b2c2);
    grid.sync();
    //========= Fc1 ============
    Fc128Layer_new(bfc1);
    grid.sync();
    //========== Output ===========
    Out128Layer_new(bout);
}


int main()
{
    int dev = 0;
    hipSetDevice(dev);
    const unsigned batch = 128;
    const unsigned output_size = 1000;
    const unsigned image_height = 224;
    const unsigned image_width = 224;
    const unsigned image_channel = 3;

    //=============== Get Input and Label =================
    float* images = (float*)malloc(batch*image_height*image_width*image_channel*sizeof(float));
    unsigned* image_labels = (unsigned*)malloc(batch*sizeof(unsigned));
//     read_ImageNet_normalized("./imagenet_files.txt", images, image_labels, batch);

    //================ Get Weight =================
    FILE* config_file = fopen("./resnet_imagenet.csv","r");

    //================ Set Network =================
    //Layer-0
    InConv128LayerParam* bconv1 = new InConv128LayerParam("Conv1", image_height, image_width, 
            7, 7, 3, 64, batch, act_bit, w_bit, 4,4,true,1,1,true);//save residual 
    InConv128LayerParam* bconv1_gpu = bconv1->initialize(images, config_file);

    //Layer-1, basic-block-1, conv1
    Conv128LayerParam* l1b1c1 = new Conv128LayerParam("L1B1C1", bconv1->output_height, 
            bconv1->output_width, 3, 3, 64, 64, batch, act_bit, w_bit);
    Conv128LayerParam* l1b1c1_gpu = l1b1c1->initialize(config_file, bconv1->get_output_gpu());

    //Layer-1, basic-block-1, conv2
    Conv128LayerParam* l1b1c2 = new Conv128LayerParam("L1B1C2", l1b1c1->output_height, 
            l1b1c1->output_width, 3, 3, 64, 64, batch,act_bit, w_bit, 1,1,true,1,1,false,true,true,64);
    Conv128LayerParam* l1b1c2_gpu = l1b1c2->initialize(config_file, l1b1c1->get_output_gpu(),
            bconv1->get_output_residual_gpu());

    //Layer-1, basic-block-2, conv1
    Conv128LayerParam* l1b2c1 = new Conv128LayerParam("L1B2C1", l1b1c2->output_height, 
            l1b1c2->output_width, 3, 3, 64, 64, batch, act_bit, w_bit);
    Conv128LayerParam* l1b2c1_gpu = l1b2c1->initialize(config_file, l1b1c2->get_output_gpu());

    //Layer-1, basic-block-2, conv2
    Conv128LayerParam* l1b2c2 = new Conv128LayerParam("L1B2C2", l1b2c1->output_height, 
            l1b2c1->output_width, 3, 3, 64, 64, batch,act_bit, w_bit, 1,1,true,1,1,false,true,true,128);
    Conv128LayerParam* l1b2c2_gpu = l1b2c2->initialize(config_file, l1b2c1->get_output_gpu(),
            l1b1c2->get_output_residual_gpu());

    //=============
    //Layer-2, basic-block-1, conv1
    Conv128LayerParam* l2b1c1 = new Conv128LayerParam("L2B1C1", l1b2c2->output_height, 
            l1b2c2->output_width, 3, 3, 64, 128, batch, act_bit, w_bit, 2, 2);
    Conv128LayerParam* l2b1c1_gpu = l2b1c1->initialize(config_file, l1b2c2->get_output_gpu());

    //Layer-2, basic-block-1, conv2
    Conv128LayerParam* l2b1c2 = new Conv128LayerParam("L2B1C2", l2b1c1->output_height, 
            l2b1c1->output_width, 3, 3, 128, 128, batch,act_bit, w_bit, 1,1,true,1,1,false,true,true,128,true);
    Conv128LayerParam* l2b1c2_gpu = l2b1c2->initialize(config_file, l2b1c1->get_output_gpu(),
            l1b2c2->get_output_residual_gpu());

    //Layer-2, basic-block-2, conv1
    Conv128LayerParam* l2b2c1 = new Conv128LayerParam("L2B2C1", l2b1c2->output_height, 
            l2b1c2->output_width, 3, 3, 128, 128, batch, act_bit, w_bit, 1, 1);
    Conv128LayerParam* l2b2c1_gpu = l2b2c1->initialize(config_file, l2b1c2->get_output_gpu());

    //Layer-2, basic-block-2, conv2
    Conv128LayerParam* l2b2c2 = new Conv128LayerParam("L2B2C2", l2b2c1->output_height, 
            l2b2c1->output_width, 3, 3, 128, 128, batch,act_bit, w_bit, 1,1,true,1,1,false,true,true,128);
    Conv128LayerParam* l2b2c2_gpu = l2b2c2->initialize(config_file, l2b2c1->get_output_gpu(),
            l2b1c2->get_output_residual_gpu());

    //=============
    //Layer-3, basic-block-1, conv1
    Conv128LayerParam* l3b1c1 = new Conv128LayerParam("L3B1C1", l2b2c2->output_height, 
            l2b2c2->output_width, 3, 3, 128, 256, batch, act_bit, w_bit, 2, 2);
    Conv128LayerParam* l3b1c1_gpu = l3b1c1->initialize(config_file, l2b2c2->get_output_gpu());

    //Layer-3, basic-block-1, conv2
    Conv128LayerParam* l3b1c2 = new Conv128LayerParam("L3B1C2", l3b1c1->output_height, 
            l3b1c1->output_width, 3, 3, 256, 256, batch,act_bit, w_bit, 1,1,true,1,1,false,true,true,128,true);
    Conv128LayerParam* l3b1c2_gpu = l3b1c2->initialize(config_file, l3b1c1->get_output_gpu(),
            l2b2c2->get_output_residual_gpu());

    //Layer-3, basic-block-2, conv1
    Conv128LayerParam* l3b2c1 = new Conv128LayerParam("L3B2C1", l3b1c2->output_height, 
            l3b1c2->output_width, 3, 3, 256, 256, batch, act_bit, w_bit, 1, 1);
    Conv128LayerParam* l3b2c1_gpu = l3b2c1->initialize(config_file, l3b1c2->get_output_gpu());

    //Layer-3, basic-block-2, conv2
    Conv128LayerParam* l3b2c2 = new Conv128LayerParam("L3B2C2", l3b2c1->output_height, 
            l3b2c1->output_width, 3, 3, 256, 256, batch,act_bit, w_bit, 1,1,true,1,1,false,true,true,256);
    Conv128LayerParam* l3b2c2_gpu = l3b2c2->initialize(config_file, l3b2c1->get_output_gpu(),
            l3b1c2->get_output_residual_gpu());

    //=============
    //Layer-4, basic-block-1, conv1
    Conv128LayerParam* l4b1c1 = new Conv128LayerParam("L4B1C1", l3b2c2->output_height, 
            l3b2c2->output_width, 3, 3, 256, 512, batch, act_bit, w_bit, 2, 2);
    Conv128LayerParam* l4b1c1_gpu = l4b1c1->initialize(config_file, l3b2c2->get_output_gpu());

    //Layer-4, basic-block-1, conv2
    Conv128LayerParam* l4b1c2 = new Conv128LayerParam("L4B1C2", l4b1c1->output_height, 
            l4b1c1->output_width, 3, 3, 512, 512, batch,act_bit, w_bit, 1,1,true,1,1,false,true,true,256,true);
    Conv128LayerParam* l4b1c2_gpu = l4b1c2->initialize(config_file, l4b1c1->get_output_gpu(),
            l3b2c2->get_output_residual_gpu());

    //Layer-4, basic-block-2, conv1
    Conv128LayerParam* l4b2c1 = new Conv128LayerParam("L4B2C1", l4b1c2->output_height, 
            l4b1c2->output_width, 3, 3, 512, 512, batch, act_bit, w_bit, 1, 1);
    Conv128LayerParam* l4b2c1_gpu = l4b2c1->initialize(config_file, l4b1c2->get_output_gpu());

    //Layer-4, basic-block-2, conv2
    Conv128LayerParam* l4b2c2 = new Conv128LayerParam("L4B2C2", l4b2c1->output_height, 
            l4b2c1->output_width, 3, 3, 512, 512, batch,act_bit, w_bit, 1,1,true,1,1,true,false,true,512);
    Conv128LayerParam* l4b2c2_gpu = l4b2c2->initialize(config_file, l4b2c1->get_output_gpu(),
            l4b1c2->get_output_residual_gpu());

    //=============
    //Layer-5
    Fc128LayerParam* bfc1 = new Fc128LayerParam("Fc1", batch, (l4b2c2->output_height)
            *(l4b2c2->output_width)*512, 512, act_bit, w_bit); 
    Fc128LayerParam* bfc1_gpu = bfc1->initialize(config_file, l4b2c2->get_output_gpu());
    //Out Layer
    Out128LayerParam* bout = new Out128LayerParam("Fout", batch, 512, output_size, act_bit, w_bit);
    Out128LayerParam* bout_gpu = bout->initialize(config_file, bfc1->get_output_gpu());  

    //================ Setup Kernel =================
    int numThreads = 128;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    int numBlocksPerSm;
    int shared_memory = 512*sizeof(int)*32;
    hipFuncSetAttribute(reinterpret_cast<const void*>(resnet128), hipFuncAttributeMaxDynamicSharedMemorySize,shared_memory);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, resnet128, numThreads, shared_memory);

    void* args[] = {&bconv1_gpu, 
        &l1b1c1_gpu, 
        &l1b1c2_gpu,
        &l1b2c1_gpu,
        &l1b2c2_gpu,
        &l2b1c1_gpu, 
        &l2b1c2_gpu,
        &l2b2c1_gpu,
        &l2b2c2_gpu,
        &l3b1c1_gpu, 
        &l3b1c2_gpu,
        &l3b2c1_gpu,
        &l3b2c2_gpu,
        &l4b1c1_gpu, 
        &l4b1c2_gpu,
        &l4b2c1_gpu,
        &l4b2c2_gpu,
        &bfc1_gpu,
        &bout_gpu};

    START_TIMER;
    for (int i=0; i<NUM_PROF; i++)
    hipLaunchCooperativeKernel((void*)resnet128, numBlocksPerSm*deviceProp.multiProcessorCount, 
            numThreads, args, shared_memory);

    CUDA_CHECK_KERNEL();
    STOP_TIMER;
    printf("Time: %.3f (ms)\n", milliseconds/NUM_PROF);

    //================ Output =================
//     float* output = bout->download_output();
    //validate_prediction(output, image_labels, output_size, batch);


/*
    float* out = l1b2c1->download_full_output();
    //float* out = l1b1c2->download_full_output();
    //for (int i=0; i<512; i++)
    for (int i=4096; i<4096+512; i++)
    {
        printf("%.f ", out[i]);
        if ((i+1)%32==0) printf("\n");
    }
    printf("\n===%f===\n", bout->bn_scale[0]);
*/ 

    delete bconv1;
    delete l1b1c1;
    delete l1b1c2;
    delete l1b2c1;
    delete l1b2c2;

    delete l2b1c1;
    delete l2b1c2;
    delete l2b2c1;
    delete l2b2c2;

    delete l3b1c1;
    delete l3b1c2;
    delete l3b2c1;
    delete l3b2c2;

    delete l4b1c1;
    delete l4b1c2;
    delete l4b2c1;
    delete l4b2c2;

    delete bfc1;
    delete bout;

    return 0;
}