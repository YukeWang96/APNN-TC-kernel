#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------
// File: alexnet.cu
// AlexNet BNN inference source file for ImageNet. 
// ---------------------------------------------------------------------------
// See our arXiv paper for detail: https://arxiv.org/abs/2006.16578
// Ang Li, Scientist, Pacific Northwest National Laboratory(PNNL), U.S.
// Homepage: http://www.angliphd.com
// GitHub repo: http://www.github.com/pnnl/TCBNN
// PNNL-IPID: 31925-E, ECCN: EAR99, IR: PNNL-SA-152850
// BSD Lincese.
// Richland, 99352, WA, USA. June-30-2020.
// ---------------------------------------------------------------------------

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <iostream>
#include <string>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <fstream>
#include <vector>
#include "utility.h"
#include "param.h"
#include "kernel.cuh"
#include "data.h"

using namespace cooperative_groups;
using namespace std;

#define NUM_PROF 100
#define w1_a2
// #define w2_a2
// #define w1_a4

#ifdef w1_a2
const unsigned w_bit = 1;
const unsigned act_bit = 2;
// #include "51_APCONV_w1a2_t64.cu"        // 0.074 ms
// #include "52_APCONV_w1a2_t128.cu"       // 0.057 ms

#include "31_APMM_w1a2_t128.cu" 
#endif

#ifdef w2_a2
const unsigned w_bit = 2;
const unsigned act_bit = 2;
// #include "48_APCONV_w2a2_t64.cu"         // 0.050 ms
// #include "49_APCONV_w2a2_t128.cu"        // 0.063 ms
#include "50_APCONV_w2a2_t128.cu"        // 0.063 ms

#include "13_APMM_w2a2_t64.cu"  //0.013 ms
// #include "29_APMM_w2a2_t128.cu"    //0.018 ms
// #include "30_APMM_w2a2_t128.cu" //0.008 ms
#endif

#ifdef w1_a4
const unsigned w_bit = 1;
const unsigned act_bit = 4;
// #include "53_APCONV_w1a4_t64.cu"            // 0.041 ms
#include "54_APCONV_w1a4_t128.cu"        // 0.098 ms only for batch-size >= 8

// #include "32_APMM_w1a4_t128.cu"           
#include "33_APMM_w1a4_t64.cu"        
#endif

#ifdef w1_a8
const unsigned w_bit = 1;
const unsigned act_bit = 8;
// #include "55_APCONV_w1a8_t64.cu"        //  ms
// #include "56_APCONV_w1a8_t128.cu"        // ms
#endif

__global__ void alexnet128(
        InConv128LayerParam* bconv1, 
        Conv128LayerParam* bconv2, 
        Conv128LayerParam* bconv3,
        Conv128LayerParam* bconv4, 
        Conv128LayerParam* bconv5, 
        Fc128LayerParam* bfc1, 
        Fc128LayerParam* bfc2, 
        Out128LayerParam* bout)
{
    grid_group grid = this_grid();
    //========= Conv1 ============
    InConv128Layer(bconv1);
    grid.sync();
    //========= Conv2 ============
    Conv128Layer(bconv2);
    grid.sync();
    //========= Conv3 ============
    Conv128Layer(bconv3);
    grid.sync();
    //========= Conv4 ============
    Conv128Layer(bconv4);
    grid.sync();
    //========= Conv5 ============
    Conv128Layer(bconv5);
    grid.sync();
    //========= Fc1 ============
    Fc128Layer(bfc1);
    grid.sync();
    //========= Fc2 ============
    Fc128Layer(bfc2);
    grid.sync();
    //========== Output ===========
    Out128Layer(bout);
}

int main()
{
    int dev = 0;
    hipSetDevice(dev);

    const unsigned batch = 128;
    const unsigned output_size = 1000;
    const unsigned image_height = 224;
    const unsigned image_width = 224;
    const unsigned image_channel = 3;
    const unsigned n_hidden = 4096;

    //=============== Get Input and Label =================
    float* images = (float*)malloc(batch*image_height*image_width*image_channel*sizeof(float));
    unsigned* image_labels = (unsigned*)malloc(batch*sizeof(unsigned));
//     read_ImageNet_normalized("./imagenet_files.txt", images, image_labels, batch);

    //================ Get Weight =================
    FILE* config_file = fopen("./alexnet_imagenet.csv","r");

    //================ Set Network =================
    //Bconv1 Layer
    InConv128LayerParam* bconv1 = new InConv128LayerParam("Conv1", image_height, image_width, 
            11, 11, 3, 64, batch, act_bit, w_bit, 4, 4, true, 2, 2); 
    InConv128LayerParam* bconv1_gpu = bconv1->initialize(images, config_file);

    //Bconv2 Layer
    Conv128LayerParam* bconv2 = new Conv128LayerParam("Conv2", bconv1->output_height, 
            bconv1->output_width, 5, 5, 64, 192, batch, act_bit, w_bit, 1, 1,
            true, 2, 2);
    Conv128LayerParam* bconv2_gpu = bconv2->initialize(config_file, bconv1->get_output_gpu());

    //Bconv3 Layer
    Conv128LayerParam* bconv3 = new Conv128LayerParam("Conv3", bconv2->output_height, 
            bconv2->output_width, 3, 3, 192, 384, batch, act_bit, w_bit);
    Conv128LayerParam* bconv3_gpu = bconv3->initialize(config_file, bconv2->get_output_gpu());

    //Bconv4 Layer
    Conv128LayerParam* bconv4 = new Conv128LayerParam("Conv4", bconv3->output_height, 
            bconv3->output_width, 3, 3, 384, 256, batch, act_bit, w_bit);
    Conv128LayerParam* bconv4_gpu = bconv4->initialize(config_file, bconv3->get_output_gpu());

    //Bconv5 Layer
    Conv128LayerParam* bconv5 = new Conv128LayerParam("Conv5", bconv4->output_height, 
            bconv4->output_width, 3, 3, 256, 256, batch, act_bit, w_bit, 1, 1, true, 2, 2, true);
    Conv128LayerParam* bconv5_gpu = bconv5->initialize(config_file, bconv4->get_output_gpu());

    //Fc1 Layer
    Fc128LayerParam* bfc1 = new Fc128LayerParam("Fc1", batch, (bconv5->output_height)
            *(bconv5->output_width)*256, n_hidden, act_bit, w_bit); 
    Fc128LayerParam* bfc1_gpu = bfc1->initialize(config_file, bconv5->get_output_gpu());
    
    //Fc2 Layer
    Fc128LayerParam* bfc2 = new Fc128LayerParam("Fc2", batch, n_hidden, n_hidden, act_bit, w_bit); 
    Fc128LayerParam* bfc2_gpu = bfc2->initialize(config_file, bfc1->get_output_gpu());
    //Out Layer
    Out128LayerParam* bout = new Out128LayerParam("Fout", batch, n_hidden, output_size, act_bit, w_bit);
    Out128LayerParam* bout_gpu = bout->initialize(config_file, bfc2->get_output_gpu());  

    //================ Setup Kernel =================
    int numThreads = 128;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    int numBlocksPerSm;
    int shared_memory = 512*sizeof(int)*32;
    hipFuncSetAttribute(reinterpret_cast<const void*>(alexnet128), hipFuncAttributeMaxDynamicSharedMemorySize,shared_memory);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, alexnet128, numThreads, shared_memory);

    void* args[] = {&bconv1_gpu, &bconv2_gpu, &bconv3_gpu, &bconv4_gpu, &bconv5_gpu, 
        &bfc1_gpu, &bfc2_gpu, &bout_gpu};


    START_TIMER;
    for (int i=0; i<NUM_PROF; i++)
    hipLaunchCooperativeKernel((void*)alexnet128, numBlocksPerSm*deviceProp.multiProcessorCount, 
            numThreads, args, shared_memory);
    CUDA_CHECK_KERNEL();
    STOP_TIMER;
    printf("Time: %.3f (ms)\n", milliseconds/NUM_PROF);

    //================ Output =================
//     float* output = bout->download_output();
    //validate_prediction(output, image_labels, output_size, batch);

    /*
    float* out = bconv2->download_full_output();
    //for (int i=0; i<512; i++)
    for (int i=4096; i<4096+512; i++)
    {
        printf("%.f ", out[i]);
        if ((i+1)%32==0) printf("\n");
    }
    printf("\n===%f===\n", bout->bn_scale[0]);
    */

    delete bconv1;
    delete bconv2;
    delete bconv3;
    delete bconv4;
    delete bconv5;
    delete bfc1;
    delete bfc2;
    delete bout;

    return 0;
}
