#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


char* concat(const char *s1, const char *s2)
{
    char *result = (char*)malloc(strlen(s1) + strlen(s2) + 1); // +1 for the null-terminator
    // in real code you would check for errors in malloc here
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void run(char * name){
	char * file_name = concat(name, ".cubin");

	int *A;
	hipMalloc((void**)&A, sizeof(char)*128*1024*1024);

	hipModule_t module;
	hipFunction_t kernel;

	hipModuleLoad(&module, file_name);
	hipModuleGetFunction(&kernel, module, "kern");

	void * args[1] = {&A};
	hipModuleLaunchKernel(kernel, 1, 1, 1,
			32, 1, 1,
			32*1024, 0, args, 0);

}

int main(){
	run("basic_LDG");
	return 0;
}
